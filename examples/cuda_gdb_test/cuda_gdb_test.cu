#include "hip/hip_runtime.h"
#include <iostream>

#include "legion.h"

using namespace Legion;

static constexpr int WarpSize = 32;

enum TaskID {
  TOP_LEVEL_ID,
  CUDA_ID
};

__device__ inline int lane_id()
{
  int id;
  asm("mov.s32 %0, %laneid;" : "=r"(id));
  return id;
}

template <typename T>
__device__ inline T warp_reduce(T val)
{
#pragma unroll
  for (int i = WarpSize / 2; i > 0; i >>= 1) {
    T tmp = __shfl_sync(val, lane_id() + i, WarpSize);
    val += tmp;
  }
  return val;
}

template <typename T, int TPB>
__device__ T block_reduce(T val, T* smem) {
  static_assert(TPB % WarpSize == 0, "Threads per block must be multiple of warp size");
  int lane = lane_id();
  int warp = threadIdx.x / WarpSize;
  T sum    = warp_reduce(val);
  if (TPB > WarpSize) {
    if (lane == 0) { smem[warp] = sum; }
    __syncthreads();
    if (warp == 0) {
      val = lane < TPB / WarpSize ? smem[lane] : T(0);
      sum = warp_reduce(val);
    }
  }
  return sum;
}

template <int TPB>
__global__ void cuda_kernel() {
  if (threadIdx.x <= 1)
    printf("I am thread %d:%d\n", blockIdx.x, threadIdx.x);
  float val = 1.f;
  __shared__ float tmp[WarpSize];
  float out = block_reduce<float, TPB>(val, &*tmp);
  if (threadIdx.x == 0)
    printf("Block: %d, reduced value: %f\n", blockIdx.x, out);
}

void cuda_task(const Task *task,
               const std::vector<PhysicalRegion> &regions,
               Context ctx, Runtime *runtime)
{
  static constexpr int TPB = 4 * WarpSize;
  dim3 block(TPB, 1);
  dim3 grid(8, 1);
  cuda_kernel<TPB><<<grid, block>>>();
}

void top_level_task(const Task *task,
                    const std::vector<PhysicalRegion> &regions,
                    Context ctx, Runtime *runtime)
{
  TaskLauncher launcher(CUDA_ID, TaskArgument(nullptr, 0));
  Future cuda_future = runtime->execute_task(ctx, launcher);
  cuda_future.wait();
}

int main(int argc, char **argv)
{
  Runtime::set_top_level_task_id(TOP_LEVEL_ID);
  {
    TaskVariantRegistrar registrar(TOP_LEVEL_ID, "top_level");
    registrar.add_constraint(ProcessorConstraint(Processor::LOC_PROC));
    Runtime::preregister_task_variant<top_level_task>(registrar, "top_level");
  }
  {
    TaskVariantRegistrar registrar(CUDA_ID, "cuda_task");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    Runtime::preregister_task_variant<cuda_task>(registrar, "cuda_task");
  }

  return Runtime::start(argc, argv);
}
